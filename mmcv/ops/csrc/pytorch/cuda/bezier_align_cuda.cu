#include "pytorch_cuda_helper.hpp"
#include "bezier_align_cuda_kernel.cuh"

void BezierAlignForwardCUDAKernelLauncher(Tensor input, Tensor beziers, Tensor output,
                                          int pooled_height, int pooled_width,
                                          float spatial_scale) {
  int output_size = output.numel();
  int channels = input.size(1);
  int height = input.size(2);
  int width = input.size(3);

  at::cuda::CUDAGuard device_guard(input.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      input.scalar_type(), "bezier_align_forward_cuda_kernel", [&] {
        bezier_align_forward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size, input.data_ptr<scalar_t>(),
                beziers.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(),
                pooled_height, pooled_width,
                static_cast<scalar_t>(spatial_scale),
                channels, height, width);
      });

  AT_CUDA_CHECK(hipGetLastError());
}

void BezierAlignBackwardCUDAKernelLauncher(Tensor grad_output, Tensor beziers,
                                           Tensor grad_input, int pooled_height,
                                           int pooled_width, float spatial_scale) {
  int output_size = grad_output.numel();
  int channels = grad_input.size(1);
  int height = grad_input.size(2);
  int width = grad_input.size(3);

  at::cuda::CUDAGuard device_guard(grad_output.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      grad_output.scalar_type(), "bezier_align_backward_cuda_kernel", [&] {
        bezier_align_backward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK, 0, stream>>>(
                output_size, grad_output.data_ptr<scalar_t>(),
                beziers.data_ptr<scalar_t>(), grad_input.data_ptr<scalar_t>(),
                pooled_height, pooled_width,
                static_cast<scalar_t>(spatial_scale),
                channels, height, width);
      });

  AT_CUDA_CHECK(hipGetLastError());
}
